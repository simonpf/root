// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 13/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

///////////////////////////////////////////////////////////////////
// Contains additional arithmetic functions required by the CUDA //
// neural network implementation.                                //
///////////////////////////////////////////////////////////////////

#include "TMVA/DNN/Architectures/Cuda.h"
#include "TMVA/DNN/Architectures/Cuda/Kernels.h"
#include "TMVA/DNN/Architectures/Cuda/Device.h"

namespace TMVA
{
namespace DNN
{

//____________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::Multiply(TCudaMatrix &C,
                                  const TCudaMatrix &A,
                                  const TCudaMatrix &B)
{
   int m, n, k;
   m = A.GetNrows();
   k = A.GetNcols();
   n = B.GetNcols();
   CudaDouble_t alpha = 1.0, beta = 0.0;

   // Compute C = beta * C + alpha * (A * B)
   hipblasDgemm(A.GetCublasHandle(),
               HIPBLAS_OP_N, HIPBLAS_OP_N,
               m, n, k, & alpha,
               A.GetDataPointer(), m,   // *A, lda
               B.GetDataPointer(), k,   // *B, ldb
               & beta,                  // beta
               C.GetDataPointer(), m);  // *C, ldc
}

//____________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::TransposeMultiply(TCudaMatrix & C,
                                           const TCudaMatrix & A,
                                           const TCudaMatrix & B)
{
   int m, n, k;
   k = A.GetNrows();
   m = A.GetNcols();
   n = B.GetNcols();
   CudaDouble_t alpha = 1.0, beta = 0.0;

   // Compute C = beta * C + alpha * (A^T * B)
   hipblasDgemm(A.GetCublasHandle(),
               HIPBLAS_OP_T, HIPBLAS_OP_N,
               m, n, k, & alpha,
               A.GetDataPointer(), k,     // *A, lda
               B.GetDataPointer(), k,     // *B, ldb
               & beta,                    // beta
               C.GetDataPointer(), m);    // *C, ldc
}

//____________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::Hadamard(TCudaMatrix &B,
                                  const TCudaMatrix &A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::Hadamard<<<gridDims, blockDims, 0, s>>>(B.GetDataPointer(),
                                                              A.GetDataPointer(),
                                                              A.GetNrows(),
                                                              A.GetNcols());
}

//____________________________________________________________________________
template<bool doProfiling>
CudaDouble_t TCuda<doProfiling>::Sum(const TCudaMatrix &A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(A);
   hipStream_t s = A.GetComputeStream();

   TCudaMatrix::ResetDeviceReturn();
   ::TMVA::DNN::Cuda::ReduceMatrix<<<gridDims, blockDims, 0, s>>>(
       TCudaMatrix::GetDeviceReturnPointer(),
       A.GetDataPointer(),
       A.GetNrows(),
       A.GetNcols());
   return TCudaMatrix::GetDeviceReturn();
}

//____________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::SumColumns(TCudaMatrix &B, const TCudaMatrix &A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(A);

   hipMemset(B.GetDataPointer(), 0, A.GetNcols() * sizeof(CudaDouble_t));
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::SumColumns<<<gridDims, blockDims, 0, s>>>(B.GetDataPointer(),
                                                                A.GetDataPointer(),
                                                                A.GetNrows(),
                                                                A.GetNcols());
}

//____________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::ScaleAdd(TCudaMatrix &B,
                                  const TCudaMatrix &A,
                                  CudaDouble_t alpha)
{
   hipblasDaxpy(A.GetCublasHandle(), A.GetNoElements(), &alpha,
               A.GetDataPointer(), 1,
               B.GetDataPointer(), 1);
}

} // DNN
} // TMVA
