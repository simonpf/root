#include "hip/hip_runtime.h"
// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 14/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

 /////////////////////////////////////////////////////////////
 // Implementation of the initialization functions for CUDA //
 // Architectures                                           //
 /////////////////////////////////////////////////////////////

#include "TRandom.h"
#include "TMVA/DNN/Architectures/Cuda.h"

namespace TMVA
{
namespace DNN
{

//______________________________________________________________________________
void TCuda::InitializeGauss(TCudaMatrix & A)
{
   size_t m,n;
   m = A.GetNrows();
   n = A.GetNcols();

   TRandom rand(time(nullptr));

   Real_t sigma = sqrt(2.0 / ((Real_t) n));

   for (size_t i = 0; i < m; i++) {
      for (size_t j = 0; j < n; j++) {
         A(i,j) = rand.Gaus(0.0, sigma);
      }
   }
}

//______________________________________________________________________________
void TCuda::InitializeUniform(TCudaMatrix & A)
{
   size_t m,n;
   m = A.GetNrows();
   n = A.GetNcols();

   TRandom rand(time(nullptr));

   Real_t range = sqrt(2.0 / ((Real_t) n));

   for (size_t i = 0; i < m; i++) {
      for (size_t j = 0; j < n; j++) {
         A(i,j) = rand.Uniform(-range, range);
      }
   }
}

//______________________________________________________________________________
void TCuda::InitializeIdentity(TCudaMatrix & A)
{
   size_t m,n;
   m = A.GetNrows();
   n = A.GetNcols();

   for (size_t i = 0; i < m; i++) {
      for (size_t j = 0; j < n ; j++) {
         A(i,j) = 0.0;
      }

      if (i < n) {
         A(i,i) = 1.0;
      }
   }
}

//______________________________________________________________________________
void TCuda::InitializeZero(TCudaMatrix & A)
{
   size_t m,n;
   m = A.GetNrows();
   n = A.GetNcols();

   for (size_t i = 0; i < m * n; i++) {
      for (size_t j = 0; j < n ; j++) {
         A(i,j) = 0.0;
      }
   }
}

} // namespace DNN
} // namespace TMVA
