#include "hip/hip_runtime.h"
// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 14/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

 /////////////////////////////////////////////////////////////
 // Implementation of the initialization functions for CUDA //
 // Architectures                                           //
 /////////////////////////////////////////////////////////////

#include "TRandom.h"
#include "TMatrix.h"
#include "TMVA/DNN/Architectures/Cuda.h"

namespace TMVA
{
namespace DNN
{

//______________________________________________________________________________
void TCuda::InitializeGauss(TCudaMatrix & A)
{
   size_t m,n;
   m = A.GetNrows();
   n = A.GetNcols();

   TRandom rand(time(nullptr));
   TMatrixT<CudaDouble_t> B(m, n);

   Real_t sigma = sqrt(2.0 / ((Real_t) n));

   for (size_t i = 0; i < m; i++) {
      for (size_t j = 0; j < n; j++) {
         B(i,j) = rand.Gaus(0.0, sigma);
      }
   }
   A = B;
}

//______________________________________________________________________________
void TCuda::InitializeUniform(TCudaMatrix & A)
{
   size_t m,n;
   m = A.GetNrows();
   n = A.GetNcols();

   TRandom rand(time(nullptr));
   TMatrixT<CudaDouble_t> B(m, n);

   Real_t range = sqrt(2.0 / ((Real_t) n));

   for (size_t i = 0; i < m; i++) {
      for (size_t j = 0; j < n; j++) {
         B(i,j) = rand.Uniform(-range, range);
      }
   }
   A = B;
}

//______________________________________________________________________________
void TCuda::InitializeIdentity(TCudaMatrix & A)
{
   size_t m,n;
   m = A.GetNrows();
   n = A.GetNcols();
   TMatrixT<CudaDouble_t> B(m, n);

   for (size_t i = 0; i < m; i++) {
      for (size_t j = 0; j < n ; j++) {
         B(i,j) = 0.0;
      }

      if (i < n) {
         B(i,i) = 1.0;
      }
   }
   A = B;
}

//______________________________________________________________________________
void TCuda::InitializeZero(TCudaMatrix & A)
{
   size_t m,n;
   m = A.GetNrows();
   n = A.GetNcols();
   TMatrixT<CudaDouble_t> B(m, n);

   for (size_t i = 0; i < m * n; i++) {
      for (size_t j = 0; j < n ; j++) {
         B(i,j) = 0.0;
      }
   }
   A = B;
}

} // namespace DNN
} // namespace TMVA
