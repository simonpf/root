// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 13/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

//////////////////////////////////////////////////////////////////////
// Implementation of the loss functions for the TCuda implementation //
// of the low-level interface.                                      //
//////////////////////////////////////////////////////////////////////

#include "TMVA/DNN/Architectures/Cuda.h"
#include "TMVA/DNN/Architectures/Cuda/Device.h"
#include "TMVA/DNN/Architectures/Cuda/Kernels.h"

namespace TMVA
{
namespace DNN
{

//____________________________________________________________________________
CudaDouble_t TCuda::MeanSquaredError(const TCudaMatrix & Y,
                                    const TCudaMatrix & output)
{
    dim3 blockDims = TDevice::BlockDims();
    dim3 gridDims  = TDevice::GridDims(Y);
    hipStream_t s = Y.GetComputeStream();
    TCudaMatrix::ResetDeviceReturn();
    mean_squared_error<<<gridDims, blockDims, 0, s>>>(
        TCudaMatrix::GetDeviceReturnPointer(),
        Y.GetDataPointer(),
        output.GetDataPointer(),
        (int) Y.GetNrows(),
        (int) Y.GetNcols());
    return TCudaMatrix::GetDeviceReturn();
}

//____________________________________________________________________________
void TCuda::MeanSquaredErrorGradients(TCudaMatrix & dY,
                                    const TCudaMatrix & Y,
                                    const TCudaMatrix & output)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(Y);
   hipStream_t s = Y.GetComputeStream();
   mean_squared_error_gradients<<<gridDims, blockDims, 0, s>>>(
       dY.GetDataPointer(),
       Y.GetDataPointer(),
       output.GetDataPointer(),
       (int) Y.GetNrows(),
       (int) Y.GetNcols());
}

//____________________________________________________________________________
CudaDouble_t TCuda::CrossEntropy(const TCudaMatrix & Y,
                                const TCudaMatrix & output)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(Y);
   TCudaMatrix::ResetDeviceReturn();
   hipStream_t s = Y.GetComputeStream();
   cross_entropy<<<gridDims, blockDims, 0, s>>>(
       TCudaMatrix::GetDeviceReturnPointer(),
       Y.GetDataPointer(),
       output.GetDataPointer(),
       (int) Y.GetNrows(),
       (int) Y.GetNcols());
   return TCudaMatrix::GetDeviceReturn();
}

//____________________________________________________________________________
void TCuda::CrossEntropyGradients(TCudaMatrix & dY,
                                 const TCudaMatrix & Y,
                                 const TCudaMatrix & output)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(Y);
   hipStream_t s = Y.GetComputeStream();
   cross_entropy_gradients<<<gridDims, blockDims, 0, s>>>(dY.GetDataPointer(),
                                                          Y.GetDataPointer(),
                                                          output.GetDataPointer(),
                                                          (int) Y.GetNrows(),
                                                          (int) Y.GetNcols());
}

} // namespace DNN
} // namespace TMVA
