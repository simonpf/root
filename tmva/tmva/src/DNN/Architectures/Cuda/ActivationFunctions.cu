// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 13/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

 //////////////////////////////////////////////////////////////////
 // Implementation of the activation functions for the TCuda      //
 // implementation of the low-level interface.                   //
 //////////////////////////////////////////////////////////////////

#include "TMVA/DNN/Architectures/Cuda.h"
#include "TMVA/DNN/Architectures/Cuda/Device.h"
#include "TMVA/DNN/Architectures/Cuda/Kernels.h"

namespace TMVA
{
namespace DNN
{

//______________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::IdentityDerivative(TCudaMatrix & A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(A);
   hipStream_t s = A.GetComputeStream();
   tick();
   ::TMVA::DNN::Cuda::IdentityDerivative<<<gridDims, blockDims, 0, s>>>(
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   tock(fTimings.TimeIdentityDerivative);
}

//______________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::Relu(TCudaMatrix & A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(A);
   hipStream_t s = A.GetComputeStream();
   tick();
   ::TMVA::DNN::Cuda::Relu<<<gridDims, blockDims, 0, s>>>(A.GetDataPointer(),
                                                  (int) A.GetNrows(),
                                                  (int) A.GetNcols());
   tock(fTimings.TimeRelu);
}

//______________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::ReluDerivative(TCudaMatrix & B,
                                        const TCudaMatrix & A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(B);
   hipStream_t s = A.GetComputeStream();
   tick();
   ::TMVA::DNN::Cuda::ReluDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   tock(fTimings.TimeReluDerivative);
}

//______________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::Sigmoid(TCudaMatrix & A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(A);
   hipStream_t s = A.GetComputeStream();
   tick();
   ::TMVA::DNN::Cuda::Sigmoid<<<gridDims, blockDims, 0, s>>>(
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
}

//______________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::SigmoidDerivative(TCudaMatrix & B,
                                           const TCudaMatrix & A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(B);
   hipStream_t s = A.GetComputeStream();
   tick();
   ::TMVA::DNN::Cuda::SigmoidDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   tock(fTimings.TimeSigmoidDerivative);
}

//______________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::Tanh(TCudaMatrix & A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(A);
   hipStream_t s = A.GetComputeStream();

   tick();
   ::TMVA::DNN::Cuda::Tanh<<<gridDims, blockDims, 0, s>>>(
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   tock(fTimings.TimeTanh);
}

//______________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::TanhDerivative(TCudaMatrix & B,
                                        const TCudaMatrix & A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(B);
   hipStream_t s = A.GetComputeStream();

   tick();
   ::TMVA::DNN::Cuda::TanhDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   tock(fTimings.TimeTanhDerivative);
}

//______________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::SymmetricRelu(TCudaMatrix & A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(A);
   hipStream_t s = A.GetComputeStream();
   tick();
   ::TMVA::DNN::Cuda::SymmetricRelu<<<gridDims, blockDims, 0, s>>>(A.GetDataPointer(),
                                                             (int) A.GetNrows(),
                                                             (int) A.GetNcols());
   tock(fTimings.TimeSymmetricRelu);
}

//______________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::SymmetricReluDerivative(TCudaMatrix & B,
                                                 const TCudaMatrix & A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(B);
   hipStream_t s = A.GetComputeStream();
   tick();
   ::TMVA::DNN::Cuda::SymmetricReluDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   tock(fTimings.TimeSymmetricReluDerivative);
}

//______________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::SoftSign(TCudaMatrix & A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(A);
   hipStream_t s = A.GetComputeStream();
   tick();
   ::TMVA::DNN::Cuda::SoftSign<<<gridDims, blockDims, 0, s>>>(
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   tock(fTimings.TimeSoftSign);
}

//______________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::SoftSignDerivative(TCudaMatrix & B, const TCudaMatrix & A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(B);
   hipStream_t s = A.GetComputeStream();

   tick();
   ::TMVA::DNN::Cuda::SoftSignDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   tock(fTimings.TimeSoftSignDerivative);
}

//______________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::Gauss(TCudaMatrix & A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(A);
   hipStream_t s = A.GetComputeStream();

   tick();
   ::TMVA::DNN::Cuda::Gauss<<<gridDims, blockDims, 0, s>>>(
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   tock(fTimings.TimeGauss);
}

//______________________________________________________________________________
template<bool doProfiling>
void TCuda<doProfiling>::GaussDerivative(TCudaMatrix & B, const TCudaMatrix & A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(B);
   hipStream_t s = A.GetComputeStream();
   tick();
   ::TMVA::DNN::Cuda::GaussDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   tock(fTimings.TimeGaussDerivative);
}

} // namespace DNN
} // namespace TMVA
