// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 11/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

////////////////////////////////////////////////////////////////
// Explicit instantiation of the Reference architecture class //
// template for Double_t scalar types.                        //
////////////////////////////////////////////////////////////////

#include "TMVA/DNN/Architectures/Cuda.h"
#include "TMVA/DNN/Architectures/Cuda/Device.h"

namespace TMVA
{
namespace DNN
{

void TCuda::Sigmoid(TCudaMatrix & B,
                    const TCudaMatrix & A)
{
   dim3 blockDims = TDevice::BlockDims();
   dim3 gridDims  = TDevice::GridDims(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::Sigmoid<<<gridDims, blockDims, 0, s>>>(B.GetDataPointer(),
                                                             A.GetDataPointer(),
                                                             (int) A.GetNrows(),
                                                             (int) A.GetNcols());
}

} // namespace DNN
} // namespace TMVA
